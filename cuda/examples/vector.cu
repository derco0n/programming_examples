// This is an example on how to calculate numbers in parallel on a nvidia GPU using Cuda.
// Compile with: nvcc vector.cu -o cuda_vector-example


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

// Kernel function to add the elements of two arrays
__global__
/// @brief This will add the values (at the index that matches the current thread id) of a and b and store it in c
/// @param a Pointer to the a-array
/// @param b Pointer to the b-array
/// @param c Pointer to the c-array
void vectorAdd(int *a, int *b, int *c)
{    
    int i= threadIdx.x; //Get the ID of the current Thread. We're using this as an index here....
    c[i] = a[i] + b[i];

    return;
}

int main(void)
{
    // Initialize input arrays
    int a[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20};
    int b[] = {21, 22, 23, 24, 25, 26, 27, 28, 29, 30, 31, 32, 33, 34, 35, 36, 37, 38, 39, 40};

    // Initialize output array
    int c[sizeof(a)/sizeof(int)]={0};

    printf("Creating pointers into the GPU...\n");
    //create pointer into the gpu
    int *cudaA =0;
    int *cudaB =0;
    int *cudaC =0;

    printf("Allocating GPU-Memory...\n");
    //allocate memory in the GPU that are of the same size as the arrays on the host (we need to copy them over later)
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    printf("Copying data to the GPU-Memory...\n");
    //copy the vectors into the gpu (Host => GPU)
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

    printf("Prforming parallel calculations...\n");
    //Calling the function that should be run in parallel like: 1vectorAdd <<< GRID_SIZE, BLOCK_SIZE >>> (ARGUMENTS)
    vectorAdd <<< 1, (sizeof(a) / sizeof(int)) >>> (cudaA, cudaB, cudaC); //Blocks: 1, Threads per Block: (sizeof(a) / sizeof(int))

    printf("Copying results from GPU's Memory to Host's memory...\n");
    //Copying the results back from the GPU into the host's memory (GPU => Host)
    hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

    printf("Printing results...\n");
    for(int res: c){
        printf("%d\n",res);
    }

    return 0;
}
